#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdint.h>
#include <cuda/barrier>
#include <hip/hip_cooperative_groups.h>
#include <cooperative_groups/memcpy_async.h>
namespace cg = cooperative_groups;

__global__ void testcopy2(float *global1, float *global2, int subset_count)
{
    extern __shared__ float shared[];
    auto group = cooperative_groups::this_thread_block();

    // Create a synchronization object 
    __shared__ cuda::barrier<cuda::thread_scope::thread_scope_block> barrier;
    if (group.thread_rank() == 0)
    {
        init(&barrier, group.size());
    }
    group.sync();

    for (size_t subset = 0; subset < subset_count; ++subset)
    {
        cuda::memcpy_async(group, shared,
                           &global1[subset * group.size()], sizeof(float) * group.size(), barrier);
        cuda::memcpy_async(group, shared + group.size(),
                           &global2[subset * group.size()], sizeof(float) * group.size(), barrier);

        barrier.arrive_and_wait(); // Wait for all copies to complete

        // simulate compute
        if (group.thread_rank() == 0)
        {
            printf("%f ", shared[0]);
        }

        barrier.arrive_and_wait();
    }
}

/*
__global__ void testcopy(float *x, int N) {
    int tid = threadIdx.x;
    __shared__ float Tile[32];
    *reinterpret_cast<float4*>(&Tile[tid]) = *reinterpret_cast<float4*>(&x[tid*4]);
    printf("%f ", Tile[tid]);
}*/

/*
__global__ void testcopy2(float *x, int N) {
    int tid = threadIdx.x;
    __shared__ float Tile[32];
    asm volatile("cp.async.cg.shared.global [%0], [%1], %2;\n"
                :: "r"((uint32_t)__cvta_generic_to_shared(&Tile[tid])),
                "l"(&x[tid]),
                "n"(16)
            );

    printf("%f ", Tile[tid]);
}
*/

int main()
{
    const int N_DATA = 1024;
    float *x;
    hipMalloc(&x, N_DATA * sizeof(float));
    float *y;
    hipMalloc(&y, N_DATA * sizeof(float));

    dim3 gridDim(32, 1, 1);
    dim3 blockDim(32, 1, 1);
    testcopy2<<<gridDim, blockDim>>>(x, y, N_DATA);

    hipFree(x);
    hipDeviceReset();

    return 0;
}
