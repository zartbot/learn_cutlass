#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>         // CUtensormap
#include <cuda/barrier>
using barrier = cuda::barrier<cuda::thread_scope_block>;
namespace cde = cuda::device::experimental;

__global__ void kernel(const CUtensorMap tensor_map, int x, int y) {

  const int tid = threadIdx.x;
  // bluk tensor 的拷贝操作需要 Shared Memory 首地址对齐 128 字节。
  __shared__ alignas(128) int smem_buffer[128][128];

  // 创建 Shared Memory 的 cuda::barrier 变量 
  #pragma nv_diag_suppress static_var_with_dynamic_init
  __shared__ barrier bar;

  if (threadIdx.x == 0) {
    // 初始化 barrier 
    init(&bar, blockDim.x);
    // 插入 fence
    cde::fence_proxy_async_shared_cta();    
  }
  __syncthreads();

  barrier::arrival_token token;
  if (threadIdx.x == 0) {
    // 发起 TMA 二维异步拷贝操作
    cde::cp_async_bulk_tensor_2d_global_to_shared(&smem_buffer, &tensor_map, x, y, bar);
    // 设置同步等待点，指定需要等待的拷贝完成的字节数。
    token = cuda::device::barrier_arrive_tx(bar, 1, sizeof(smem_buffer));
  } else {
    // Other threads just arrive.
    token = bar.arrive();
  }
  // 等待完成拷贝
  bar.wait(std::move(token));

  smem_buffer[0][threadIdx.x] += threadIdx.x;

  // 插入 fence
  cde::fence_proxy_async_shared_cta();
  __syncthreads();

  if (threadIdx.x == 0) {
    cde::cp_async_bulk_tensor_2d_shared_to_global(&tensor_map, x, y, &smem_buffer);
    cde::cp_async_bulk_commit_group();
    cde::cp_async_bulk_wait_group_read<0>();
  }

  if (threadIdx.x == 0) {
    (&bar)->~barrier();
  }
}