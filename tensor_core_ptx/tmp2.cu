
#include <hip/hip_runtime.h>
#include <hip/hip_fp16.h>
#include <mma.h>
using namespace nvcuda;

__global__ void test_wmma(half  *C, half *A, half *B)
{
        wmma::fragment<wmma::matrix_a, 16, 16, 16, __half, wmma::row_major> a_frag;
        wmma::fragment<wmma::matrix_b, 16, 16, 16, __half, wmma::col_major> b_frag;

//        wmma::fragment<wmma::matrix_a, 16, 16, 16, __half, wmma::col_major> a_frag;
//        wmma::fragment<wmma::matrix_b, 16, 16, 16, __half, wmma::row_major> b_frag;
        

        wmma::fragment<wmma::accumulator, 16, 16, 16, __half> acc_frag;

        wmma::load_matrix_sync( a_frag, A, 16 );
        wmma::load_matrix_sync( b_frag, B, 16 );
        wmma::fill_fragment( acc_frag, 0.0f );
        
        wmma::mma_sync( acc_frag, a_frag, b_frag, acc_frag );
        wmma::store_matrix_sync( C, acc_frag, 16, wmma::mem_row_major );
}

//  nvcc -c -arch sm_70 --ptx  tmp2.cu 
// nvcc -c -arch sm_70 tmp2.cu ; cuobjdump -sass tmp2.o > tmp.sass

