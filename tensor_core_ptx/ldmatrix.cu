
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdint.h>
#include "hip/hip_fp16.h"
// #include "mma.h"

#define WARP_SIZE 32

#define LDMATRIX_X1(R, addr) \
    asm volatile("ldmatrix.sync.aligned.x1.m8n8.shared.b16 {%0}, [%1];\n" : "=r"(R) : "r"(addr))

#define LDMATRIX_X2(R0, R1, addr) \
    asm volatile("ldmatrix.sync.aligned.x2.m8n8.shared.b16 {%0, %1}, [%2];\n" : "=r"(R0), "=r"(R1) : "r"(addr))

#define LDMATRIX_X4(R0, R1, R2, R3, addr)                                             \
    asm volatile("ldmatrix.sync.aligned.x4.m8n8.shared.b16 {%0, %1, %2, %3}, [%4];\n" \
                 : "=r"(R0), "=r"(R1), "=r"(R2), "=r"(R3)                             \
                 : "r"(addr))

#define LDMATRIX_X1T(R, addr) \
    asm volatile("ldmatrix.sync.aligned.x1.trans.m8n8.shared.b16 {%0}, [%1];\n" : "=r"(R) : "r"(addr))

#define LDMATRIX_X2T(R0, R1, addr) \
    asm volatile("ldmatrix.sync.aligned.x2.trans.m8n8.shared.b16 {%0, %1}, [%2];\n" : "=r"(R0), "=r"(R1) : "r"(addr))

#define LDMATRIX_X4T(R0, R1, R2, R3, addr)                                                  \
    asm volatile("ldmatrix.sync.aligned.x4.trans.m8n8.shared.b16 {%0, %1, %2, %3}, [%4];\n" \
                 : "=r"(R0), "=r"(R1), "=r"(R2), "=r"(R3)                                   \
                 : "r"(addr))

__global__ void TestLDMatrix(void)
{
    const int tid = threadIdx.x;

    __shared__ uint16_t M[4 * 16 * 16];
    if (tid == 0)
    {
        int offset = 0;
        for (int i = 0; i < 4; ++i){
            for (int j = 0; j < 16; ++j){
                for (int k = 0; k < 16; ++k)
                {
                    M[offset] = static_cast<uint16_t>((i+1) * 10000 + (j+1) * 100 + k+1);
                    printf(" %6d",M[offset]);
                    offset++;
                }
                printf("\n");
            }
             printf("\n");
        }
    }

    __syncthreads();

    int offset = tid * 32;

    uint32_t addr = __cvta_generic_to_shared(M + offset);

    uint32_t frag[4];
    //LDMATRIX_X1(frag[0],addr);
    LDMATRIX_X4T(frag[0], frag[1], frag[2], frag[3], addr);
    uint16_t data[4][2];
    for (int i = 0; i < 4; ++i)
    {
        data[i][0] = static_cast<uint16_t>(frag[i] & 0xFFFF);
        data[i][1] = static_cast<uint16_t>((frag[i] >> 16) & 0xFFFF);
    }
    printf("OFFSET %4d  tid: %3d | A | %6d %6d | %6d %6d | %6d %6d | %6d %6d |\n", offset, tid,
           int(data[0][0]), int(data[0][1]), int(data[1][0]), int(data[1][1]),
           int(data[2][0]), int(data[2][1]), int(data[3][0]), int(data[3][1]));
}

int main(void)
{
    dim3 gridDim(1, 1, 1);
    dim3 blockDim(32, 1, 1);
    TestLDMatrix<<<gridDim, blockDim>>>();

    hipDeviceReset();

    return 0;
}


        /*
    int aTile_index = tid % 16 * 8 + tid / 16 * 8;

        const uint32_t address =
            cvta_to_shared_u32(M) + sizeof(uint16_t) * ((tid%8) * 8);*/

