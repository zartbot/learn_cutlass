#include <getopt.h>
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <cute/tensor.hpp>

using namespace cute;

#define MAXN  128*128

int main()
{
    // initial memory with physical layout
    int* A = (int*)malloc(MAXN * sizeof(int));
    for(int i =0 ; i < MAXN ; i++){
	    A[i]=int(i);
    }   

    auto sa = make_shape(Int<2>{},Int<2>{});
    auto a = make_layout(sa, Stride<_1,_6>{});
    Tensor ta =make_tensor(A, a);
    print_tensor(ta);

    auto c = complement(a, 24);
    Tensor tc =make_tensor(A, c);
    print_tensor(tc);
   
}