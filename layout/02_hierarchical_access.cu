

#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <cute/tensor.hpp>
#include <cutlass/numeric_types.h>

using namespace cute;

#define PRINT_LAYOUT(name, content) \
    print(name);                    \
    print(" : ");                   \
    print(content);                 \
    print(" Shape: ");              \
    print(cute::shape(content));    \
    print(" Stride: ");             \
    print(cute::stride(content));   \
    print(" rank: ");               \
    print(cute::rank(content));     \
    print(" depth: ");              \
    print(cute::depth(content));    \
    print(" size: ");               \
    print(cute::size(content));     \
    print(" cosize: ");             \
    print(cute::cosize(content));   \
    print("\n");

int main()
{

    auto s1 = make_shape(_1{}, _2{});
    auto d1 = make_stride(_1{}, _2{});
    auto s2 = make_shape(_2{}, _3{}, s1);
    auto d2 = make_stride(_2{}, _3{}, d1);
    auto s3 = make_shape(_3{}, _4{}, _5{}, s2);
    auto d3 = make_stride(_3{}, _4{}, _5{}, d2);
    auto s4 = make_shape(_4{}, _5{}, _6{}, s3);
    auto d4 = make_stride(_4{}, _5{}, _6{}, d3);
    auto s5 = make_shape(_5{}, _6{}, _7{},_8{}, s4);
    auto d5 = make_stride(_5{}, _6{}, _7{},_8{}, d4);

    Layout a = make_layout(s5, d5);
    PRINT_LAYOUT("a", a);
    PRINT_LAYOUT("a<4>", get<4>(a));
    auto a43 = get<4,3>(a);
    PRINT_LAYOUT("a<4,3>",a43 );
    auto a433 = get<4,3,3>(a);
    PRINT_LAYOUT("a<4,3,3>",a433 );
    auto a4332 = get<4,3,3,2>(a);
    PRINT_LAYOUT("a<4,3,3,2> ",a4332 );
    auto a4332_1 = get<2>(a433);
    PRINT_LAYOUT("a<4,3,3,2>1",a4332_1 );
}
