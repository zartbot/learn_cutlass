#include <getopt.h>
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <cute/tensor.hpp>

using namespace cute;

#define MAXN 128 * 128

int main()
{
    // initial memory with physical layout
    int *A = (int *)malloc(MAXN * sizeof(int));
    for (int i = 0; i < MAXN; i++)
    {
        A[i] = int(i);
    }

    auto layout_a = make_layout(make_shape(Int<4>{}, Int<2>{}),
                                make_stride(Int<1>{}, Int<16>{}));
    Tensor ta = make_tensor(A, layout_a);
    printf("\nLayout Tensor A: ");
    print_tensor(ta);

    auto layout_b= make_layout(make_shape(Int<6>{}),
                                make_stride(Int<1>{}));
    // auto layout_b = make_layout(make_shape(Int<2>{}, Int<4>{}),
    //                            make_stride(Int<4>{}, Int<2>{}));
    Tensor tb = make_tensor(A, layout_b);
    printf("\nLayout Tensor B: ");
    print_tensor(tb);

    Layout a_star = complement(layout_a, size(layout_a) * cosize(layout_b));
    Tensor ta_star = make_tensor(A, a_star);
    printf("\nLayout Tensor A* : ");
    print_tensor(ta_star);

    Layout a_star2 = composition(complement(layout_a, size(layout_a) * cosize(layout_b)), layout_b);
    Tensor ta_star2 = make_tensor(A, a_star2);
    printf("\nLayout Tensor A* o B: ");
    print_tensor(ta_star2);

    auto lp = logical_product(layout_a, layout_b);

    Tensor tlp = make_tensor(A, lp);
    printf("\nLayout Tensor Logical Product: ");
    print_tensor(tlp);
}

/*

// B-Tile < 3:3, (2,4):(1:8) >
    auto tiler = make_tile(Layout<_3, _3>{},     // Apply     3:3     to mode-0
                           Layout<Shape<_2, _4>, // Apply (2,4):(1,8) to mode-1
                                  Stride<_1, _8>>{});

                                     printf("\nLayout Tensor Logical Divide(mode-0): ");
    print_tensor(tensor<0>(tld));


    // ((TileM,TileN), (RestM,RestN)) with shape ((3,8), (3,4))
    auto zd = zipped_divide(layout_a, tiler);

    Tensor tzd = make_tensor(A, zd);
    printf("\nLayout Tensor Zipped Divide: ");
    print_tensor(tzd);
    printf("\nLayout Tensor Zipped Divide(mode-0): ");
    print_tensor(tensor<0>(tzd));
*/