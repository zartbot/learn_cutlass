

#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <cute/tensor.hpp>
#include <cutlass/numeric_types.h>

using namespace cute;

#define PRINT(name, content)      \
    print(name);                  \
    print(" : ");                 \
    print(content);               \
    print(" Shape: ");            \
    print(cute::shape(content));  \
    print(" Stride: ");           \
    print(cute::stride(content)); \
    print(" rank: ");             \
    print(cute::rank(content));   \
    print(" depth: ");            \
    print(cute::depth(content));  \
    print(" size: ");             \
    print(cute::size(content));   \
    print(" cosize: ");           \
    print(cute::cosize(content)); \
    print("\n");


int main()
{

    Layout a = make_layout(make_shape(_6{}, _2{}), make_stride(_1{}, _7{}));
    Layout b = make_layout(make_shape(_3{}, _2{}), make_stride(_2{}, _3{}));
    Layout c = composition(a, b);
    Layout d = complement(a, c);
    Layout e = make_layout(a, c);

    PRINT("a", a);
    PRINT("b", b);
    PRINT("c", c);
    PRINT("c-get<1>", get<1>(c));
    PRINT("d", d);
    PRINT("e", e);

    Layout f_col = make_layout(make_shape(Int<2>{},3,4,5,6),
                               LayoutLeft{});
    Layout f_row = make_layout(make_shape(Int<2>{},3,4,5,6),
                               LayoutRight{});
    PRINT("fcol", f_col);
    PRINT("frow", f_row);
                  
    
}
