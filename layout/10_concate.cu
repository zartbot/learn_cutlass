#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <cute/tensor.hpp>

using namespace cute;

int main()
{
    Layout a = Layout<_3,_1>{};                     // 3:1
    Layout b = Layout<_4,_3>{};                     // 4:3

    Layout row = make_layout(a, b);                 // (3,4):(1,3)
    printf("\nrow = make_layout(a, b) :");
    print(row);

    Layout col = make_layout(b, a);                 // (4,3):(3,1)
    printf("\ncol = make_layout(b, a) :");
    print(col);

    Layout q   = make_layout(row, col);             // ((3,4),(4,3)):((1,3),(3,1))
    printf("\nq = make_layout(row, col); :");
    print(q);

    auto sc = Shape<_3,_4,_5>();
    Layout c = make_layout(sc,LayoutLeft{});
    printf("\nc :");
    print(c);
   
    auto sd = Shape<_6,_7,_8>();
    Layout d = make_layout(sd,LayoutLeft{});
    printf("\nd :");
    print(d);

    Layout cd = make_layout(sc,sd);
    printf("\nmake_layout(c,d); :");
    print(cd);
    

Layout aa  = make_layout(a);                    // (3):(1)
Layout aaa = make_layout(aa);                   // ((3)):((1))


}



