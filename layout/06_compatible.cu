#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <cute/tensor.hpp>

#define MAXN 128*128

using namespace cute;


template<class T1,class T2>
void print_compatible(T1 l1, T2 l2) {
    print(l1);
    printf(" -> ");
    print(l2);
    printf(" is ");
    if (is_compatible<decltype(l1),decltype(l2)>()) {
        printf("compatible\n");
    } else {
        printf("NOT compatible\n");
    }
}


int main()
{
    
    auto s1 = make_shape(_24{});
    
    printf("reflexive\n");
    print_compatible(s1,s1);

    printf("\n\ntransitive\n");
    auto s3 = make_shape(make_tuple(_4{},_6{}));    
    auto s5 = make_shape(make_tuple(make_tuple(_2{},_2{}),_6{}));
    print_compatible(s1,s3);
    print_compatible(s3,s5);
    print_compatible(s1,s5);

    printf("\n\nantisymetric\n");
    auto s2 = make_shape(make_tuple(_24{}));
    print_compatible(s1,s2);
    print_compatible(s2,s1);
    print_compatible(s1,s3);
    print_compatible(s3,s1);

    printf("\n\nothers\n");
    auto s4 = make_shape(make_tuple(_2{},_3{}),_4{});
    auto s6 = make_shape(make_tuple(_2{},_3{},_4{}));
    print_compatible(s1,s4);
    print_compatible(s1,s6);


    auto s7 = make_shape(make_tuple(make_tuple(_2{},_3{}),_4{}));
    print_compatible(s1,s7);
    print_compatible(s3,s7);






}
