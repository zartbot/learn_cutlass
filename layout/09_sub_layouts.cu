#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <cute/tensor.hpp>

using namespace cute;

int main()
{
    Layout a =  Layout<Shape<_4,Shape<Shape<_4,_5>,Shape<_6,_7>>>>{}; 
    printf("a :");
    print(a);

    printf("\nlayout<0>(a) :");
    print(layout<0>(a));

    printf("\nlayout<1>(a) :");
    print(layout<1>(a));

    printf("\nlayout<1,0>(a) :");
    print(layout<1,0>(a));

    printf("\nlayout<1,1>(a) :");
    print(layout<1,1>(a));

    printf("\nlayout<1,1,0>(a) :");
    print(layout<1,1,0>(a));

    Layout b= Layout<Shape<_2,_3,_5,_7>>{};  
    printf("\nb :");
    print(b);

    printf("\nselect<2>(b) :");
    print(select<2>(b));

    printf("\nselect<1,3>(b) :");
    print(select<1,3>(b));

    printf("\nselect<0,1,3>(b) :");
    print(select<0,1,3>(b));

    printf("\ntake<1,3>(b) :");
    print(take<1,3>(b));

    printf("\ntake<1,4>(b) :");
    print(take<1,4>(b));


}


Layout a = Layout<_3,_1>{};                     // 3:1
Layout b = Layout<_4,_3>{};                     // 4:3
Layout row = make_layout(a, b);                 // (3,4):(1,3)
Layout col = make_layout(b, a);                 // (4,3):(3,1)
Layout q   = make_layout(row, col);             // ((3,4),(4,3)):((1,3),(3,1))
Layout aa  = make_layout(a);                    // (3):(1)
Layout aaa = make_layout(aa);                   // ((3)):((1))
Layout d   = make_layout(a, make_layout(a), a); // (3,(3),3):(1,(1),1)