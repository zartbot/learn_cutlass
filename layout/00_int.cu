#include "hip/hip_runtime.h"

#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <cute/tensor.hpp>
#include <cutlass/numeric_types.h>

using namespace cute;



#define PRINT_TUPLE(name, content)      \
    print(name);                  \
    print(" : ");                 \
    print(content);               \
    print(" rank: ");             \
    print(cute::rank(content));   \
    print(" depth: ");            \
    print(cute::depth(content));  \
    print(" size: ");             \
    print(cute::size(content));   \
    print("\n");


int main()
{
    //动态整型
    auto dynamic_var = int{2};
    dynamic_var = 4;

    bool is_dynamic = cute::is_integral<decltype(dynamic_var)>();
    printf("%d \n",is_dynamic);
    
    //静态整型
    auto static_var = Int<3>{};
    is_dynamic = cute::is_integral<decltype(static_var)>();
    printf("%d \n",is_dynamic);  
    // static_var  -= 3 , compile error

    
    //复合运算
    auto var = Int<8>{} + static_var + max (_4{}, _3{}) - abs(_m4{}) * dynamic_var;
    printf ("var= %d \n",var);

    auto a =  make_tuple(uint16_t{42}, int{7});
    PRINT_TUPLE("a",a);
    auto b =  make_tuple(uint16_t{4}, int{8},Int<9>{} );
    PRINT_TUPLE("b",b);
    auto c = make_tuple(uint16_t{42}, make_tuple(Int<1>{}, int32_t{3}), b);
    PRINT_TUPLE("c",c);
    PRINT_TUPLE("c",get<2>(c));


}