#include <getopt.h>
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <cute/tensor.hpp>

using namespace cute;

#define MAXN 128 * 128
template <class T>
auto make_composition(T a, int N, int r)
{
    auto shape_b = make_shape(N);
    auto stride_b = make_stride(r);
    Layout b = make_layout(shape_b, stride_b);
    printf("\nLayout-B: ");
    print(b);
    auto c = composition(a, b);
    printf("\nLayout-A o B: ");
    print(c);
    printf("\n");
    return c;
}

int main()
{


    auto sa = make_shape(Int<36>{}, Int<18>{});
    auto a = make_layout(sa, make_stride(Int<1>{}, Int<72>{}));

    auto sb = make_shape(Int<9>{}, Int<4>{});
    auto b = make_layout(sb, make_stride(Int<4>{}, Int<9>{}));
    
    auto c = composition(a, b);
    print(c);







    auto s2 = make_shape(Int<4>{}, Int<6>{}, Int<8>{},Int<10>{});
    auto a2 = make_layout(s2, make_stride(Int<2>{}, Int<3>{}, Int<5>{},Int<7>{}));
    auto b2 = make_layout(make_shape(Int<6>{}), make_stride(Int<12>{}));
    auto c2 = composition(a2, b2);
    print(c2);


    
}