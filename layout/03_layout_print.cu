#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <cute/tensor.hpp>

using namespace cute;

template <class Shape, class Stride>
void print2D(Layout<Shape, Stride> const &layout)
{
    for (int m = 0; m < size<0>(layout); ++m)
    {
        for (int n = 0; n < size<1>(layout); ++n)
        {
            printf("%3d  ", layout(m, n));
        }
        printf("\n");
    }
}

int main()
{

    Layout s46_col = make_layout(make_shape(Int<4>{}, 6),
                                   LayoutLeft{});
    Layout s46_row = make_layout(make_shape(Int<4>{}, 6),
                                   LayoutRight{});
/*
    printf("2d-col-major layout\n");
    print2D(s46_col);
    printf("2d-row-major layout\n");
    print2D(s46_row);

    print_layout(s46_col);

    print_latex(s46_col);

    */
   auto coord = make_coord(2,3);
   int inner_product = (int)get<0>(coord)  * (int)stride<0>(s46_col) +  
                       (int)get<1>(coord)  * (int)stride<1>(s46_col) ;
   printf("%3d %3d\n",s46_col(coord), s46_col(inner_product));

}