#include <getopt.h>
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <cute/tensor.hpp>

using namespace cute;

#define MAXN 128 * 128

int main()
{
    // initial memory with physical layout
    int *A = (int *)malloc(MAXN * sizeof(int));
    for (int i = 0; i < MAXN; i++)
    {
        A[i] = int(i);
    }

    // A: shape is (9,32)
    auto layout_a = make_layout(make_shape(Int<9>{}, make_shape(Int<4>{}, Int<8>{})),
                                make_stride(Int<59>{}, make_stride(Int<13>{}, Int<1>{})));
    Tensor ta = make_tensor(A, layout_a);
    printf("\nLayout Tensor A: ");
    print_tensor(ta);

    // B-Tile < 3:3, (2,4):(1:8) >
    auto tiler = make_tile(Layout<_3, _3>{},     // Apply     3:3     to mode-0
                           Layout<Shape<_2, _4>, // Apply (2,4):(1,8) to mode-1
                                  Stride<_1, _8>>{});

    // ((TileM,RestM), (TileN,RestN)) with shape ((3,3), (8,4))
    auto ld = logical_divide(layout_a, tiler);

    Tensor tld = make_tensor(A, ld);
    printf("\nLayout Tensor Logical Divide: ");
    print_tensor(tld);
     printf("\nLayout Tensor Logical Divide(mode-0): ");
    print_tensor(tensor<0>(tld));


    // ((TileM,TileN), (RestM,RestN)) with shape ((3,8), (3,4))
    auto zd = zipped_divide(layout_a, tiler);

    Tensor tzd = make_tensor(A, zd);
    printf("\nLayout Tensor Zipped Divide: ");
    print_tensor(tzd);
    printf("\nLayout Tensor Zipped Divide(mode-0): ");
    print_tensor(tensor<0>(tzd));
}