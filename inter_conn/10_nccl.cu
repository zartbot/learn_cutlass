

#include <hip/hip_runtime.h>
#include <nccl.h>
#include <cstdio>
#include <cstdlib>
#include <time.h>
 
__global__ void kernel(int *a) 
{
  int index = threadIdx.x;

  a[index] *= 2;
  printf("%d\t", a[index]);

}/*kernel*/
 

void print_vector(int *in, int n){

 for(int i=0; i < n; i++)
  printf("%d\t", in[i]);

 printf("\n");

}/*print_vector*/


int main(int argc, char* argv[]) {

  srand(time(NULL)); 
  int data_size = 16384;
  int nGPUs = 0;
  hipGetDeviceCount(&nGPUs);
  
  int *DeviceList = (int *) malloc (nGPUs     * sizeof(int));
  int *data       = (int*)  malloc (data_size * sizeof(int));
  int **d_data    = (int**) malloc (nGPUs     * sizeof(int*));
  
  for(int i = 0; i < nGPUs; i++)
      DeviceList[i] = i;
  
  /*Initializing NCCL with Multiples Devices per Thread*/
  ncclComm_t* comms = (ncclComm_t*)  malloc(sizeof(ncclComm_t)  * nGPUs);  
  hipStream_t* s   = (hipStream_t*)malloc(sizeof(hipStream_t)* nGPUs);
  ncclCommInitAll(comms, nGPUs, DeviceList);
  
  /*Population the data vector*/
  for(int i = 0; i < data_size; i++)
      data[i] = rand()%(20);
 
  print_vector(data, data_size);
      
  for(int g = 0; g < nGPUs; g++) {
      hipSetDevice(DeviceList[g]);
      hipStreamCreate(&s[g]);
      hipMalloc(&d_data[g], data_size * sizeof(int));
     
      if(g == 0)  /*Copy from Host to Device*/
         hipMemcpy(d_data[g], data, data_size * sizeof(int), hipMemcpyHostToDevice);
  }
        
  ncclGroupStart();
  for (int i = 0 ; i < 1024000 ; ++i ) {
 
  		for(int g = 0; g < nGPUs; g++) {
  	  	    hipSetDevice(DeviceList[g]);
    	  	    ncclBcast(d_data[g], data_size, ncclInt, 0, comms[g], s[g]); /*Broadcasting it to all*/
  		}
  }

  ncclGroupEnd();    


  for(int g = 0; g < nGPUs; g++)    /*Finalizing NCCL*/
     ncclCommDestroy(comms[g]);
  
  /*Freeing memory*/
  free(s);
  free(data); 
  free(DeviceList);

  hipFree(d_data);

  return 0;

}/*main*/


/*

  for (int g = 0; g < nGPUs; g++) {
      cudaSetDevice(DeviceList[g]);
      printf("\nThis is device %d\n", g);
      kernel <<< 1 , data_size >>> (d_data[g]);//Call the CUDA Kernel: The code multiple the vector position per 2 on GPUs
      cudaDeviceSynchronize();             
  }

  printf("\n");

  for (int g = 0; g < nGPUs; g++) { //Synchronizing CUDA Streams
      cudaSetDevice(DeviceList[g]);
      cudaStreamSynchronize(s[g]);
  }
 
  for(int g = 0; g < nGPUs; g++) {  //Destroy CUDA Streams
      cudaSetDevice(DeviceList[g]);
      cudaStreamDestroy(s[g]);
  }
*/