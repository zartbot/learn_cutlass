/*%****************************************************************************80
%  Code: 
%   ncclSendRecv.cu
%
%  Purpose:
%   Implements sample send/recv code using the package NCCL (p2p).
%
%  Modified:
%   Aug 18 2020 10:57 
%
%  Author:
%   Murilo Boratto <murilo.boratto 'at' fieb.org.br>
%
%  How to Compile:
%   nvcc ncclSendRecv.cu -o object -lnccl  
%
%  HowtoExecute: 
%   ./object 
%                         
%****************************************************************************80*/


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <nccl.h>

__global__ void kernel(int *a, int rank) { 

  if(rank == 0)
    printf("%d\t", a[threadIdx.x]); 
      else
        printf("%d\t", a[threadIdx.x]*10); 
}
 
void show_all(int *in, int n){

 printf("\n");

 for(int i=0; i < n; i++)
    printf("%d\t", in[i]);
      
 printf("\n");

}/*show_all*/


int main(int argc, char* argv[]) {

  int size = 4096;

  /*Get current amounts number of GPU*/
  int nGPUs = 0;
  hipGetDeviceCount(&nGPUs);
  printf("nGPUs = %d\n",nGPUs);

  /*List GPU Device*/
  int *DeviceList = (int *) malloc ( nGPUs * sizeof(int));

  for(int i = 0; i < nGPUs; ++i)
      DeviceList[i] = i;
  
  /*NCCL Init*/
  ncclComm_t* comms         = (ncclComm_t*)  malloc(sizeof(ncclComm_t)  * nGPUs);  
  hipStream_t* s           = (hipStream_t*)malloc(sizeof(hipStream_t)* nGPUs);
  ncclCommInitAll(comms, nGPUs, DeviceList); 

  /*General variables*/
  int *host       = (int*) malloc(size      * sizeof(int));
  int **sendbuff  = (int**)malloc(nGPUs     * sizeof(int*));
  int **recvbuff  = (int**)malloc(nGPUs     * sizeof(int*));
  
  /*Population of vector*/
  for(int i = 0; i < size; i++)
      host[i] = i + 1;

  show_all(host, size);

  for(int g = 0; g < nGPUs; g++) {
      hipSetDevice(DeviceList[g]);
      hipStreamCreate(&s[g]);
      hipMalloc(&sendbuff[g], size * sizeof(int));
      hipMalloc(&recvbuff[g], size * sizeof(int));
     
      if(g == 0)
        hipMemcpy(sendbuff[g], host, size * sizeof(int),hipMemcpyHostToDevice);
       
  }/*for*/
  
  ncclGroupStart();        
    for (int i = 0 ; i < 10240000 ; ++i ) {

  	for(int g = 0; g < nGPUs; g++) {
          ncclSend(sendbuff[0], size, ncclInt, g, comms[g], s[g]);
    	    ncclRecv(recvbuff[g], size, ncclInt, g, comms[g], s[g]);
    }
    }
  
  ncclGroupEnd();          


  for(int g = 0; g < nGPUs; g++) {
     ncclCommDestroy(comms[g]);
  }
  
  free(s);
  free(host);
  
  hipFree(sendbuff);
  hipFree(recvbuff);

  return 0;

}


/*

  for(int g = 0; g < nGPUs; g++) {
      cudaSetDevice(DeviceList[g]);
      printf("\nThis is device %d\n", g);
      if(g==0)
      kernel <<< 1 , size >>> (sendbuff[g], g); 
      else
      kernel <<< 1 , size >>> (recvbuff[g], g); 
      cudaDeviceSynchronize();
  }

 printf("\n");

  for (int g = 0; g < nGPUs; g++) {
      cudaSetDevice(DeviceList[g]);
      cudaStreamSynchronize(s[g]);
  }

  
  for(int g = 0; g < nGPUs; g++) {
      cudaSetDevice(DeviceList[g]);
      cudaStreamDestroy(s[g]);
  }

*/