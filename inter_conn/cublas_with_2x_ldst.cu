
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>
#include <cstdint>
#include "hipblas.h"

#define M_GLOBAL 4096
#define N_GLOBAL 4096
#define K_GLOBAL 4096
#define ITER 1000

void launch_gemm(hipStream_t stream, size_t M, size_t N, size_t K, half *A, half *B, half *C, half alpha, half beta)
{
    hipblasHandle_t handle;
    hipblasCreate(&handle);
     hipblasSetStream(handle,stream);
    hipblasSetMathMode(handle, HIPBLAS_TENSOR_OP_MATH);

    hipblasGemmEx(handle, HIPBLAS_OP_T, HIPBLAS_OP_N, N, M, K, &alpha, B, HIP_R_16F, K, A,
                 HIP_R_16F, K, &beta, C, HIP_R_16F, N, HIPBLAS_COMPUTE_16F,
                 CUBLAS_GEMM_DEFAULT_TENSOR_OP);
}


__global__ void SmallKernel(float *local_mem, float *remote_mem1)
{
    const int idx = blockIdx.x * blockDim.x + threadIdx.x;
    for (int i = 0; i < 32; ++i)
    {
        remote_mem1[idx + i * 32] = local_mem[idx + i * 32] * 2.0f;
    }
   // __nanosleep(10000);
}

int main()
{
    uint32_t size = pow(2, 30); // Memory Copy Size
    const int ngpu = 2;

    float *dev[ngpu];
    for (int i = 0; i < ngpu; ++i)
    {
        hipSetDevice(0);
        hipMalloc((void **)&dev[i], size);
        for (int j = 0; j < ngpu; ++j)
        {
            if (i != j)
            {
                hipDeviceEnablePeerAccess(i, j);
            }
        }
    }
    hipSetDevice(0);

    const float alpha = 1.0f;
    const float beta = 0.0f;

    half *d_a, *d_b, *d_c;
    hipMalloc(&d_a, M_GLOBAL * K_GLOBAL * sizeof(half));
    hipMalloc(&d_b, K_GLOBAL * N_GLOBAL * sizeof(half));
    hipMalloc(&d_c, M_GLOBAL * N_GLOBAL * sizeof(half));

    const int nStreams = 4;
    int priority_high, priority_low;
    hipStream_t stream[nStreams];
    for (int i = 0; i < nStreams; i++)
    {
        //cudaStreamCreate(&stream[i]);
        //cudaStreamCreateWithPriority(&stream[i], cudaStreamNonBlocking, priority_low);
        hipStreamCreateWithFlags(&stream[i], hipStreamNonBlocking);

    }

    hipEvent_t start, end;
    hipEventCreate(&start);
    hipEventCreate(&end);

    hipEventRecord(start);
    for (int i = 0; i < ITER; i++)
    {
        launch_gemm(stream[2],M_GLOBAL, N_GLOBAL, K_GLOBAL, d_a, d_b, d_c, alpha, beta);
        // SimpleKernel<<<1, 32>>>(dev[0], dev[1], dev[2], dev[3]); // 执行GPU0 Kernel
        SmallKernel<<<1, 32, 1, stream[3]>>>(dev[0], dev[1]); 
    }

    hipEventRecord(end);
    hipEventSynchronize(end);

    float msec;
    hipEventElapsedTime(&msec, start, end);

    long workload = long(M_GLOBAL) * N_GLOBAL * K_GLOBAL * 2 * ITER;
    double avg_Gflops = ((double)workload / 1e9) / (double(msec) / 1e3);
    printf("Average Performance  %10.1lf Gflops\n", avg_Gflops);

    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

    for (int i = 0; i < nStreams; i++)
    {
        hipStreamDestroy(stream[i]);
    }

    hipDeviceSynchronize();
    for (int i = 0; i < ngpu; ++i)
    {
        hipFree(dev[i]);
    }
}