#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <cute/tensor.hpp>
using namespace cute;

int main()
{
  
    Tensor A = make_tensor<int>(make_shape(_4{},_8{}),GenColMajor{});
    fill(A, 7);
    print_tensor(A);
    clear(A);
    print_tensor(A);
    fill(A, 3);

    Tensor B = make_tensor<int>(make_shape(_4{},_8{}),GenColMajor{});
    fill(B, 2);

    //B = 3 * A + 2 * B
    axpby(3,A, 2, B);
    print_tensor(B);
}

