#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <cute/tensor.hpp>
using namespace cute;

int main()
{

    Tensor A = make_tensor<float>(Shape<Shape<_4, _5>, Int<13>>{},
                                  Stride<Stride<_12, _1>, _64>{});
    float *b_ptr = (float *)malloc(13 * 20 * sizeof(float));
    Tensor B = make_tensor(b_ptr, make_shape(13, 20));

    // Fill A via natural coordinates op[]
    for (int m0 = 0; m0 < size<0, 0>(A); ++m0)
        for (int m1 = 0; m1 < size<0, 1>(A); ++m1)
            for (int n = 0; n < size<1>(A); ++n)
                A[make_coord(make_coord(m0, m1), n)] = n + 2 * m0;

    // Transpose A into B using variadic op()
    for (int m = 0; m < size<0>(A); ++m)
        for (int n = 0; n < size<1>(A); ++n)
            B(n, m) = A(m, n);

    // Copy B to A as if they are arrays
    for (int i = 0; i < A.size(); ++i)
        A[i] = B[i];
    
    print_tensor(A);
    print_tensor(B);

    free(b_ptr);
}