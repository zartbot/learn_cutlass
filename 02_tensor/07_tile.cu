#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <cute/tensor.hpp>
using namespace cute;

#define MAXN 128 * 128

#define PRINT(name, tensor) \
    printf("%8s : ", name); \
    print_tensor(tensor);   \
    print("\n");

int main()
{
    // initial memory
    int *hA = (int *)malloc(MAXN * sizeof(int));
    for (int i = 0; i < MAXN; ++i)
        hA[i] = i;

    // (4,6):(_1,4)
    Tensor A = make_tensor(hA, make_shape(4, 6));
    PRINT("A", A)
    auto tiler = Shape<_2, _3>{};

    //((_2,_3),(2,2)):((_1,4),(_2,12))
    Tensor tiled_a = zipped_divide(A, tiler);

    // inner
    int blockIdx_x = 0;
    int blockIdx_y = 1;
    Tensor cta_a = tiled_a(make_coord(_, _), make_coord(blockIdx_x, blockIdx_y));
    PRINT("CTA_A", cta_a)
    Tensor local_tileA = local_tile(A, tiler, make_coord(0, 1));
    PRINT("LOCAL_TILE", local_tileA)

    // outer
    int threadIdx_x = 3;
    Tensor thr_a = tiled_a(threadIdx_x, make_coord(_, _));
    PRINT("THR_A", thr_a)
    Tensor outer_partA = outer_partition(A, tiler, make_coord(1, 1));
    PRINT("OUTER_PART", outer_partA)
    Tensor local_partA = local_partition(A, make_layout(Shape<_2, _3>{}), 3);
    PRINT("LOCAL_PART", local_partA)

    free(hA);
}