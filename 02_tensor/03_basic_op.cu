#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <cute/tensor.hpp>

using namespace cute;

#define MAXN 128 * 128

#define PRINT(name, tensor)  \
    printf("%20s : ", name); \
    print(tensor);           \
    print("\n");

__global__ void tensor_kernel(float *A)
{
    Tensor t = make_tensor(A, make_shape(_8{}, _4{}), GenColMajor{});
    PRINT("tensor_8x4", t)
    PRINT("Layout", t.layout())
    PRINT("SHAPE", t.shape())
    PRINT("STRIDE", t.stride())
    PRINT("SIZE", t.size())
    PRINT("Data", t.data())
    PRINT("Rank", t.rank)
    PRINT("Depth", depth(t))
}

int main()
{
    // initial memory
    float *A = (float *)malloc(MAXN * sizeof(float));
    for (int i = 0; i < MAXN; i++)
    {
        A[i] = float(i);
    }

    float *dA;
    hipMalloc(&dA, MAXN * sizeof(float));
    hipMemcpy(dA, A, MAXN * sizeof(float), hipMemcpyHostToDevice);

    tensor_kernel<<<1, 1>>>(dA);
    hipDeviceSynchronize();
    free(A);
    hipFree(dA);
}