#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <cute/tensor.hpp>

using namespace cute;

#define MAXN 128 * 128

#define PRINTTENSOR(name, tensor) \
    printf("%20s : ", name);      \
    print(tensor);                \
    print("\n");

__global__ void tensor_kernel(float *A)
{
    // Untagged pointers
    Tensor tensor_8 = make_tensor(A, make_layout(Int<8>{})); // Construct with Layout
    Tensor tensor_8s = make_tensor(A, Int<8>{});             // Construct with Shape
    Tensor tensor_8d2 = make_tensor(A, 8, 2);                // Construct with Shape and Stride
    PRINTTENSOR("tensor_8", tensor_8)
    PRINTTENSOR("tensor_8s", tensor_8s)
    PRINTTENSOR("tensor_8d2", tensor_8d2)

    // Global memory (static or dynamic layouts)
    Tensor gmem_8s = make_tensor(make_gmem_ptr(A), Int<8>{});
    Tensor gmem_8d = make_tensor(make_gmem_ptr(A), 8);
    Tensor gmem_8sx16d = make_tensor(make_gmem_ptr(A), make_shape(Int<8>{}, 16));
    Tensor gmem_8dx16s = make_tensor(make_gmem_ptr(A), make_shape(8, Int<16>{}),
                                     make_stride(Int<16>{}, Int<1>{}));
    PRINTTENSOR("gmem_8s", gmem_8s)
    PRINTTENSOR("gmem_8d", gmem_8d)
    PRINTTENSOR("gmem_8sx16d", gmem_8sx16d)
    PRINTTENSOR("gmem_8dx16s", gmem_8dx16s)

    // Shared memory (static or dynamic layouts)
    Layout smem_layout = make_layout(make_shape(Int<4>{}, Int<8>{}));
    __shared__ float smem[decltype(cosize(smem_layout))::value]; // (static-only allocation)

    Tensor smem_4x8_col = make_tensor(make_smem_ptr(smem), smem_layout);
    Tensor smem_4x8_row = make_tensor(make_smem_ptr(smem), shape(smem_layout), GenRowMajor{});
    PRINTTENSOR("smem_4x8_col", smem_4x8_col)
    PRINTTENSOR("smem_4x8_row", smem_4x8_row)
}

int main()
{
    // initial memory 
    float *A = (float *)malloc(MAXN * sizeof(float));
    for (int i = 0; i < MAXN; i++)
    {
        A[i] = float(i);
    }

    float *dA;
    hipMalloc(&dA, MAXN * sizeof(float));
    hipMemcpy(dA, A, MAXN * sizeof(float), hipMemcpyHostToDevice);

    // Untagged pointers
    Tensor tensor_8 = make_tensor(A, make_layout(Int<8>{})); // Construct with Layout
    Tensor tensor_8s = make_tensor(A, Int<8>{});             // Construct with Shape
    Tensor tensor_8d2 = make_tensor(A, 8, 2);                // Construct with Shape and Stride
    PRINTTENSOR("host_tensor_8", tensor_8)
    PRINTTENSOR("host_tensor_8s", tensor_8s)
    PRINTTENSOR("host_tensor_8d2", tensor_8d2)
    printf("\n");

    tensor_kernel<<<1, 1>>>(dA);
    hipDeviceSynchronize();
    free(A);
    hipFree(dA);
}