#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <cute/tensor.hpp>
using namespace cute;

#define MAXN 128 * 128

int main()
{
    // initial memory
    int *a_ptr = (int *)malloc(MAXN * sizeof(int));
    for (int i = 0; i < MAXN; ++i)
        a_ptr[i] = i;

    //(_3,_4,_5):(_20,_5,_1)
    Tensor A = make_tensor(a_ptr, make_shape(Int<3>{}, Int<4>{}, Int<5>{}),
                           GenRowMajor{});

    print_tensor(A);
    Tensor A1 = A(_, _, 2);
    print_tensor(A1);

    //(_3,_4),(_2,_4,_2)):((_64,_16),(_8,_2,_1)
    Tensor B = make_tensor(a_ptr, make_shape(make_shape(Int<3>{}, Int<4>{}), make_shape(Int<2>{}, Int<4>{}, Int<2>{})),
                           GenRowMajor{});

    print_tensor(B);
    Tensor C = B(make_coord(_, _), make_coord(1, 2, 1));
    print_tensor(C);

    Tensor D = B(make_coord(1, _), make_coord(0, _, 1));
    print_tensor(D);

    Tensor E = take<0,1>(B);
    print_tensor(E);

    Tensor F = take<0,1>(A);
    print_tensor(F);

}