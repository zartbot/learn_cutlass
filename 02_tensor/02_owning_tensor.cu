#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <cute/tensor.hpp>
using namespace cute;

#define PRINTTENSOR(name, tensor) \
    printf("%20s : ", name);      \
    print(tensor);                \
    print("\n");

__global__ void tensor_kernel()
{
      // Register memory (static layouts only)
    Tensor rmem_4x8_col = make_tensor<float>(Shape<_4, _8>{});
    Tensor rmem_4x8_row = make_tensor<float>(Shape<_4, _8>{},
                                             LayoutRight{});
    Tensor rmem_4x8_pad = make_tensor<float>(Shape<_4, _8>{},
                                             Stride<_32, _2>{});
    Tensor rmem_4x8_like = make_tensor_like(rmem_4x8_pad);
    PRINTTENSOR("rmem_4x8_col", rmem_4x8_col)
    PRINTTENSOR("rmem_4x8_row", rmem_4x8_row)
    PRINTTENSOR("rmem_4x8_pad", rmem_4x8_pad)
    PRINTTENSOR("rmem_4x8_like", rmem_4x8_like)
}

int main()
{
    // Register memory (static layouts only)
    Tensor rmem_4x8_col = make_tensor<float>(Shape<_4, _8>{});
    Tensor rmem_4x8_row = make_tensor<float>(Shape<_4, _8>{},
                                             LayoutRight{});
    Tensor rmem_4x8_pad = make_tensor<float>(Shape<_4, _8>{},
                                             Stride<_32, _2>{});
    Tensor rmem_4x8_like = make_tensor_like(rmem_4x8_pad);
    PRINTTENSOR("host_rmem_4x8_col", rmem_4x8_col)
    PRINTTENSOR("host_rmem_4x8_row", rmem_4x8_row)
    PRINTTENSOR("host_rmem_4x8_pad", rmem_4x8_pad)
    PRINTTENSOR("host_rmem_4x8_like", rmem_4x8_like)
    printf("\n");

    tensor_kernel<<<1, 1>>>();
    hipDeviceSynchronize();
}