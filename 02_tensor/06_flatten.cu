#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <cute/tensor.hpp>
using namespace cute;

#define MAXN 128 * 128

int main()
{
    // initial memory
    int *a_ptr = (int *)malloc(MAXN * sizeof(int));
    for (int i = 0; i < MAXN; ++i)
        a_ptr[i] = i;

    //(_3,_4),(_2,_4,_2)):((_64,_16),(_8,_2,_1)
    Tensor B = make_tensor(a_ptr, make_shape(make_shape(Int<3>{}, Int<4>{}), make_shape(Int<2>{}, Int<4>{}, Int<2>{})),
                           GenRowMajor{});

    //(_3,_4,_2,_4,_2):(_64,_16,_8,_2,_1)
    Tensor C = flatten(B);
    print_tensor(C);

    // ((_3,_4),(_2,_4,_2)):((_1,_3),(_12,_24,_96))
    Tensor D = make_tensor(a_ptr, make_shape(make_shape(Int<3>{}, Int<4>{}), make_shape(Int<2>{}, Int<4>{}, Int<2>{})),
                           GenColMajor{});
    print_tensor(D);
    //_192:_1
    Tensor E = coalesce(D);
    print_tensor(E);

    //(_3,(_4,_2,_4),_2):(_64,(_16,_8,_2),_1):
    Tensor F = group_modes<1,4>(C);
    print_tensor(F);
}