#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <cute/tensor.hpp>
using namespace cute;

#define MAXN 128 * 128

int main()
{
    // initial memory
    int *hA = (int *)malloc(MAXN * sizeof(int));
       for (int i = 0; i < MAXN; ++i)
        hA[i] = i;
  

    // Construct a TV-layout that maps 8 thread indices and 4 value indices
    //   to 1D coordinates within a 4x8 tensor
    // (T8,V4) -> (M4,N8)
    auto tv_layout = Layout<Shape<Shape<_2, _4>, Shape<_2, _2>>,
                            Stride<Stride<_8, _1>, Stride<_4, _16>>>{}; // (8,4)
    print_layout(tv_layout);

    Tensor A = make_tensor(hA,make_shape(_4{},_8{}),GenColMajor{});
    print_tensor(A);
    // Compose A with the tv_layout to transform its shape and order
    Tensor tv = composition(A, tv_layout); // (8,4)
    // Slice so each thread has 4 values in the shape and order that the tv_layout prescribes
    int tid = 1;
    Tensor v = tv(tid, _); // (4)
    print_tensor(v);
}

