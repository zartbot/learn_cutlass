#include<hip/hip_runtime.h>

__global__ void kernel(float* D, uint64_t desc_a, uint64_t desc_b, const int scaleA, const int scaleB, int scale_D, const int tnspA,const int tnspB) {
     float d[16];

     for (int i = 0 ; i < 16 ; ++i ) {
       d[i]=0;
     }
    
    asm volatile(
    "{\n"
      ".reg .pred p;\n"
      "setp.ne.b32 p, %10, 0;\n"
      "wgmma.mma_async.sync.aligned.m64n16k16.f32.f16.f16 "
      "{%0,  %1,  %2,  %3,  %4,  %5,  %6,  %7},"
      " %8,"
      " %9,"
      " p,   1, 1 , 0 , 0; \n"
    "}\n"
      : "+f"(d[0]), "+f"(d[1]), "+f"(d[2]), "+f"(d[3]),
        "+f"(d[4]), "+f"(d[5]), "+f"(d[6]), "+f"(d[7])
      :  "l"(desc_a),
         "l"(desc_b),
         "r"(int32_t(scale_D)));
    
    //防止编译器优化
    desc_a++;
    desc_b++;
    scale_D=1;

    asm volatile(
    "{\n"
      ".reg .pred p;\n"
      "setp.ne.b32 p, %10, 0;\n"
      "wgmma.mma_async.sync.aligned.m64n16k16.f32.f16.f16 "
      "{%0,  %1,  %2,  %3,  %4,  %5,  %6,  %7},"
      " %8,"
      " %9,"
      " p,   1, 1 , 0 , 0; \n"
    "}\n"
      : "+f"(d[8]), "+f"(d[9]), "+f"(d[10]), "+f"(d[11]),
        "+f"(d[12]), "+f"(d[13]), "+f"(d[14]), "+f"(d[15])
      :  "l"(desc_a),
         "l"(desc_b),
         "r"(int32_t(scale_D)));

    asm volatile("wgmma.commit_group.sync.aligned;");
    asm volatile("wgmma.wait_group.sync.aligned 0;");         

    //store to GMEM
    for(int i = 0 ; i < 16 ; ++i ) {
      D[i] = d[i];
    }
}