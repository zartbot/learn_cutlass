
#include <hip/hip_runtime.h>
#include "mma.h"
using namespace nvcuda;

__global__ void matmulT(float *C, half *A, half *B, int Ay, int Ax, int Bx)
{
    // warp rank in grid
    int warp = (blockDim.x * blockIdx.x + threadIdx.x) / warpSize;
    int cx = warp % (Bx / 16);    // (x,y) location if active tile
    int cy = warp / (Bx / 16);    // for current warp in C matrix
    int Atile_pos = cy * 16 * Bx; // start x (row) for first A tile
    int Btile_pos = cx * 16;      // start y (col) for first B tile

    // Declare the fragments as 16 x 16 tiles
    wmma::fragment<wmma::matrix_a, 16, 16, 16, half, wmma::row_major> a_frag; // A
    wmma::fragment<wmma::matrix_b, 16, 16, 16, half, wmma::row_major> b_frag; // B
    wmma::fragment<wmma::accumulator, 16, 16, 16, float> c_frag;              // C
    wmma::fill_fragment(c_frag, 0.0f);                                        // set C = 0

    // load A as 16x16 tile
    wmma::load_matrix_sync(a_frag, &A[Atile_pos], Ax);
    // load B as 16x16 tile
    wmma::load_matrix_sync(b_frag, &B[Btile_pos], Bx);
    // C = A*B + C
    wmma::mma_sync(c_frag, a_frag, b_frag, c_frag);
    wmma::store_matrix_sync(&C[(cy * Bx + cx) * 16], c_frag, Bx, wmma::mem_row_major);
}
