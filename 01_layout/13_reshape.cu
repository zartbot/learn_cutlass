#include <getopt.h>
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <cute/tensor.hpp>

using namespace cute;

#define MAXN  128*128

int main()
{
    // initial memory with physical layout
    int* A = (int*)malloc(MAXN * sizeof(int));
    for(int i =0 ; i < MAXN ; i++){
	    A[i]=int(i);
    }   

    auto sa = make_shape(Int<20>{});
    auto a = make_layout(sa, Stride<_2>{});
    Tensor ta =make_tensor(A, a);
    print_tensor(ta);

    auto sb = make_shape(Int<5>{}, Int<4>{});
    auto b = make_layout(sb, make_stride(Int<4>{}, Int<2>{}));
    Tensor tb =make_tensor(A, b);
    print_tensor(tb);
    print(cosize(b));
    
    auto c = composition(a, b);
    Tensor tc =make_tensor(A, c);
    print_tensor(tc);
   
}