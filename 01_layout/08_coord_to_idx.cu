#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <cute/tensor.hpp>

using namespace cute;

int main()
{
    auto shape = Shape<_4, Shape<_2, _4>>{};
    auto stride = Stride<_2,Stride<_1,_8>>{};
    auto l = make_layout(shape,stride);
    print_layout(l);

    printf("\ncrd2idx 22 : "); 
    print(crd2idx(22, shape, stride)); 

    printf("\ncrd2idx (2,5) : "); 
    print(crd2idx(make_coord(2,5), shape, stride)); 

    printf("\ncrd2idx (2,(1,2)) : "); 
    print(crd2idx(make_coord(2,make_coord(1,2)), shape, stride)); 

    printf("\n");

}

