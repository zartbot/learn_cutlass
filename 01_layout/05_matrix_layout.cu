#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <cute/tensor.hpp>

#define MAXN 128*128

using namespace cute;

#define PRINTTENSOR(name,  tensor) \
    printf("\nTensor : %s :",name);                 \
    print_tensor(tensor);                 \
    print("\n");      

int main()
{
    // initial memory with physical layout
    int* A = (int*)malloc(MAXN * sizeof(int));
    for(int i =0 ; i < MAXN ; i++){
	    A[i]=int(i);
    }   
    
    // 2D tensor
    auto shape2d = make_shape(_4{},_8{});

    //(_4,_8):(_1,_4)
    Layout l1 = make_layout(shape2d, LayoutLeft{});
    Tensor t1 = make_tensor(A, l1);
    PRINTTENSOR("LayoutLeft",t1)

    //(_4,_8):(_8,_1)
    Layout l2 = make_layout(shape2d, LayoutRight{});
    Tensor t2 = make_tensor(A, l2);
    PRINTTENSOR("LayoutRight",t2)
    
    //(_4,_8):(_3,_2)
    Layout l3 = make_layout(shape2d, make_stride(_3{},_2{}));
    Tensor t3 = make_tensor(A, l3);
    PRINTTENSOR("(_4,_8):(_3,_2)",t3)
}


/*

    auto layout = Layout<Shape<_16, _16>,
                       Stride<_16, _1>>{};
    auto swizzled_layout = composition(Swizzle<2,0,3>{}, layout);
    Tensor s_2d = make_tensor(A, swizzled_layout);
    PRINTTENSOR("2d swizzled_layout", swizzled_layout, s_2d)
    //print_latex(swizzled_layout);



    // 3D tensor
    auto shape3 = make_shape(Int<2>{}, Int<3>{}, Int<4>{});
    auto stride3 = make_stride(Int<12>{}, Int<4>{}, Int<1>{});
    auto layout3 = make_layout(shape3, stride3);
    Tensor t_3d = make_tensor(A,layout3);
    PRINTTENSOR("3d", layout3, t_3d);

*/