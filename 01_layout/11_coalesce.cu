#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <cute/tensor.hpp>

using namespace cute;


template<class T>
void print_coalesce(T layout) {
    printf("H-Layout  :");
    print(layout);
    printf("\nCoalesce-Layout :");
    print(coalesce(layout));
    printf("\n");
}

int main()
{
    Layout a0 = make_layout(Shape<_2,_4>{},Stride<_1,_2>{});
    print_coalesce(a0);
    
    auto s1 = Shape<_2,Shape<_3,_4>>();
    auto s2 = Shape<_5,Shape<_6,_7>>();
    auto s3 = make_shape(s1,s2);
    
    Layout a_col = make_layout(s3, GenColMajor{});  //GenColMajor == LayoutLeft
    print_coalesce(a_col);

    Layout a_row = make_layout(s3, GenRowMajor{});  //GenRowMajor == LayoutRight
    print_coalesce(a_row);

    printf("\nCoalesce-Layout :");
    auto result = coalesce(a_col, Step<_1,_1>{});   //(_24,_210):(_1,_24) 

    auto b1 = coalesce(a_col,Step<_1,Step<_1,_1>>{});
    print(b1);
}
//H-Layout a :((_2,(_3,_4)),(_5,(_6,_7))):((_1,(_2,_6)),(_24,(_120,_720)))
//Coalesce-Layout :_5040:_1

/*
    auto s1 = make_shape(_1{}, _2{});
    auto d1 = make_stride(_1{}, _2{});
    auto s2 = make_shape(_2{}, _3{}, s1);
    auto d2 = make_stride(_2{}, _3{}, d1);
    auto s3 = make_shape(_3{}, _4{}, _5{}, s2);
    auto d3 = make_stride(_3{}, _4{}, _5{}, d2);
    auto s4 = make_shape(_4{}, _5{}, _6{}, s3);
    auto d4 = make_stride(_4{}, _5{}, _6{}, d3);
    auto s5 = make_shape(_5{}, _6{}, _7{},_8{}, s4);
    auto d5 = make_stride(_5{}, _6{}, _7{},_8{}, d4);
*/