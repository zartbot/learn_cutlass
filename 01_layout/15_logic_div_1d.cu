#include <getopt.h>
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <cute/tensor.hpp>

using namespace cute;

#define MAXN  128*128

int main()
{
    // initial memory with physical layout
    int* A = (int*)malloc(MAXN * sizeof(int));
    for(int i =0 ; i < MAXN ; i++){
	    A[i]=int(i);
    }   

    //layout-a
    auto sa = make_shape(Int<4>{},Int<2>{},Int<3>{});
    auto da = make_stride(Int<2>{},Int<1>{},Int<8>{});
    auto a = make_layout(sa, da);
    Tensor ta =make_tensor(A, a);
    printf("\nLayout A: ");
    print(ta);

    //layout-a
    auto sb = make_shape(Int<4>{});
    auto db = make_stride(Int<2>{});
    auto b = make_layout(sb, db);
    Tensor tb =make_tensor(A, b);
    printf("\nLayout B: ");
    print_tensor(tb);

    auto b_star = complement(b, size(a));
    Tensor tb_star =make_tensor(A, b_star);
    printf("\nLayout B*: ");
    print_tensor(tb_star);

    auto c1 = composition(a,b);
    Tensor tc1 =make_tensor(A, c1);
    auto c2 = composition(a,b_star);
    Tensor tc2 =make_tensor(A, c2);
    printf("\nLayout A o B: ");
    print_tensor(tc1);
    printf("\nLayout A o B*: ");
    print_tensor(tc2);

    auto d = logical_divide(a,b);
    Tensor td =make_tensor(A, d);
     printf("\nLayout A div B: ");
    print_tensor(td);
  
}