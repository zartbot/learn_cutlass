#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <cute/tensor.hpp>
using namespace cute;

#define MAXN 128*128
#define PRINTTENSOR(name,  tensor) \
    print(name);                          \
    print("\nTensor : ");                 \
    print_tensor(tensor);                 \
    print("\n");                    

int main()
{

    // initial memory with physical layout
    int* A = (int*)malloc(MAXN * sizeof(int));
    for(int i =0 ; i < MAXN ; i++){
	    A[i]=int(i);
    }   

    auto shape_1d = make_shape(Int<8>{});

    //Layout _8:_1
    Tensor t_1d = make_tensor(A, make_layout(shape_1d, make_stride(_1{})));
    PRINTTENSOR("1d layout",t_1d)
    
    //Layout _8:_2
    Tensor t_s2 = make_tensor(A,make_layout(shape_1d, make_stride(_2{})));
    PRINTTENSOR("1d stride2",t_s2)
    
    //Layout _8:_m1
    Tensor t_s_m1 = make_tensor(A+7,make_layout(shape_1d, make_stride(_m1{})));
    PRINTTENSOR("1d stride -1",t_s_m1)

    //Layout _8:_m2
    Tensor t_s_m2 = make_tensor(A+16,make_layout(shape_1d, make_stride(_m2{})));
    PRINTTENSOR("1d stride -1",t_s_m2)

}

/*


    printf("Coord : ");
    for (int i = 0 ; i < shape<0>(layout_1d); ++i) {
        printf("%3d ", i);
    }
    printf("\n");
    printf("Index : ");
    for (int i = 0 ; i < shape<0>(layout_1d); ++i) {
        printf("%3d ", A[i]);
    }
    printf("\n");           // 1D tensor
    auto shape_1d = make_shape(Int<8>{});
    auto stride_1d = make_stride(Int<-1>{});
    auto layout_1d = make_layout(shape_1d, stride_1d);
    Tensor t_1d = make_tensor(A+7, layout_1d);
    PRINTTENSOR("1d layout",layout_1d,t_1d)
*/