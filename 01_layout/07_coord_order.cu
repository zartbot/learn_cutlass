#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <cute/tensor.hpp>

using namespace cute;

int main()
{
    auto shape = Shape<_3, Shape<_5, _4>>{};

    printf("\nidx2crd 19 : "); 
    print(idx2crd(19, shape)); 

    printf("\nidx2crd (1,5) : "); 
    print(idx2crd(make_coord(1, 5), shape));  
    
    printf("\nidx2crd (1,(1,2)) : "); 
    print(idx2crd(make_coord(1, make_coord(1, 2)), shape));   

    printf("\ncrd2idx (1,5) : ");
    print(crd2idx(make_coord(1, 5), shape));printf("\n"); 
}

/*

    Layout s46_col = make_layout(make_shape(Int<4>{}, 6),
                                 LayoutLeft{});
    print_layout(s46_col);
    //    print_latex(s46_col);

    for (int m = 0; m < size<0>(s46_col); ++m)
    {
        for (int n = 0; n < size<1>(s46_col); ++n)
        {
            auto coord = make_coord(m, n);
            int inner_product = (int)get<0>(coord) * (int)stride<0>(s46_col) +
                                (int)get<1>(coord) * (int)stride<1>(s46_col);
            printf("Coord[%3d,%3d]: %3d | verify %d\n", m, n, s46_col(coord),  s46_col(coord)==s46_col(inner_product));
        }
        printf("\n");
    }

    */