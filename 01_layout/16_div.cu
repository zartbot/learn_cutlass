#include <getopt.h>
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <cute/tensor.hpp>

using namespace cute;

#define MAXN  128*128

int main()
{
    /*
   Layout raked_prod = Layout<Shape <Shape < _3,_2>,Shape <_4,_2>>,
                           Stride<Stride<_16,_1>,Stride<_4,_2>>>{};
Tile   subtile    = make_tile(Layout<_2,_3>{},    // Gather elements 2 : 3 from mode 0
                              Layout<_2,_4>{});   // Gather elements 2 : 4 from mode 1

print_layout(logical_divide(raked_prod, subtile));*/

// A: shape is (9,32)
auto layout_a = make_layout(make_shape (Int< 9>{}, make_shape (Int< 4>{}, Int<8>{})),
                            make_stride(Int<59>{}, make_stride(Int<13>{}, Int<1>{})));
// B: shape is (3,8)
auto tiler = make_tile(Layout<_3,_3>{},           // Apply     3:3     to mode-0
                       Layout<Shape <_2,_4>,      // Apply (2,4):(1,8) to mode-1
                              Stride<_1,_8>>{});

// ((TileM,RestM), (TileN,RestN)) with shape ((3,3), (8,4))
auto ld = logical_divide(layout_a, tiler);
// ((TileM,TileN), (RestM,RestN)) with shape ((3,8), (3,4))
auto zd = zipped_divide(layout_a, tiler);

print_layout(ld);
printf("\n zip-div :\n");
print_layout(zd);
}